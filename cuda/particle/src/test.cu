#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "particle.h"

__global__ void advanceParticles(float dt, particle * pArray, int nParticles)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < nParticles)
    {
        pArray[idx].advance(dt);
    }
}

int main(int argc, char ** argv)
{
    hipError_t error;
    int n = 1000000;
    if(argc > 1)    { n = atoi(argv[1]);}     // Number of particles
    if(argc > 2)    {   srand(atoi(argv[2])); } // Random seed

    error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("0 %s\n",hipGetErrorString(error));
    exit(1);
    }

    particle * pArray = new particle[n];
    particle * devPArray = NULL;
    hipMalloc(&devPArray, n*sizeof(particle));
    hipDeviceSynchronize(); error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("1 %s\n",hipGetErrorString(error));
    exit(1);
    }

    hipMemcpy(devPArray, pArray, n*sizeof(particle), hipMemcpyHostToDevice);
    hipDeviceSynchronize(); error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("2 %s\n",hipGetErrorString(error));
    exit(1);
    }

    for(int i=0; i<100; i++)
    {
        float dt = (float)rand()/(float) RAND_MAX; // Random distance each step
        advanceParticles<<< 1 +  n/256, 256>>>(dt, devPArray, n);
        error = hipGetLastError();
        if (error != hipSuccess)
        {
        printf("3 %s\n",hipGetErrorString(error));
        exit(1);
        }

        hipDeviceSynchronize();
    }
    hipMemcpy(pArray, devPArray, n*sizeof(particle), hipMemcpyDeviceToHost);

    v3 totalDistance(0,0,0);
    v3 temp;
    for(int i=0; i<n; i++)
    {
        temp = pArray[i].getTotalDistance();
        totalDistance.x += temp.x;
        totalDistance.y += temp.y;
        totalDistance.z += temp.z;
    }
    float avgX = totalDistance.x /(float)n;
    float avgY = totalDistance.y /(float)n;
    float avgZ = totalDistance.z /(float)n;
    float avgNorm = sqrt(avgX*avgX + avgY*avgY + avgZ*avgZ);
    printf( "Moved %d particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n",
                    n, avgX, avgY, avgZ, avgNorm);
    return 0;
}